#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define BLOCK_SIZE 32
#define ROW_BLOCK_SIZE BLOCK_SIZE
#define COL_BLOCK_SIZE BLOCK_SIZE
#define MAX_SM_SIZE 32* BLOCK_SIZE
#define NUM_SIMULTANEOUS_C_ROW_ELEMENTS 2
#define NUM_SIMULTANEOUS_C_COL_ELEMENTS 2
#define NUM_SM_TILES 1
#define min(a,b) (((a)<(b))?(a):(b))
//128 KB shared memory/L1 cache
//13 (out of 15 actual) SMXs
//3.7 compute capability
//1024 threads/ block
//64 warps/ smx
//2048 threads/smx
//16 blocks/smx
//=>
//13 * 16 blocks, 1 grid

//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMulNaive(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    int row =  blockIdx.y*blockDim.y + threadIdx.y;
    int col =  blockIdx.x*blockDim.x + threadIdx.x;

    if((row < square_dim) && (col < square_dim)){ // if within matrix bounds
        _DOUBLE_ _c = 0;

        for (unsigned int k = 0; k < square_dim; ++k) {
            _DOUBLE_ a = A[row * square_dim + k];
            _DOUBLE_ b = B[k * square_dim + col];
            _c += a * b;//
        }

        C[row * square_dim + col] = _c;
    }

}

//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMul_UNROLL2(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {



    __shared__ _DOUBLE_ A_block[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
    __shared__ _DOUBLE_ B_block[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];

    __shared__ _DOUBLE_ A_block_1[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
    __shared__ _DOUBLE_ B_block_1[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];

    int block_row = threadIdx.y;
    int block_col = threadIdx.x;
  //deal with block size != tile size
  int block_x_offset = min(COL_BLOCK_SIZE, blockDim.x);
  int block_y_offset = min(ROW_BLOCK_SIZE, blockDim.y);

  int row =  blockIdx.y*block_y_offset + block_row;
  int col =  blockIdx.x*block_x_offset + block_col;
//    int index = row *square_dim + col;
//    int row = blockIdx.y * blockDim.y + block_col;
    _DOUBLE_ _c = 0;

    int col_offset;
    int row_offset;
    int A_index;
    int B_index;
    int num_unrolls = 2;
#pragma unroll
    for (unsigned int kk = 0; kk < gridDim.x; kk+=num_unrolls) {

        col_offset = (kk*block_x_offset + block_col);
        row_offset = (kk*block_y_offset + block_row );
        A_index = row * square_dim + col_offset;
        B_index = row_offset * square_dim + col;

        //load sub-blocks into shared memory: each thread does one load
        if (A_index < square_dim*square_dim) {
            A_block[block_row][block_col] = A[A_index];
        } else {// this thread walks off array
            A_block[block_row][block_col] = 0;
        }

        if (B_index < square_dim*square_dim) {
            B_block[block_row][block_col] = B[B_index];
        } else {// this thread walks off array
            B_block[block_row][block_col] = 0;
        }

        col_offset = ((kk+1)*block_x_offset + block_col);
        row_offset = ((kk+1)*block_y_offset + block_row );

        A_index = row * square_dim + col_offset;
        B_index = row_offset * square_dim + col;

        //load sub-blocks into shared memory: each thread does one load
        if (A_index < square_dim*square_dim) {
            A_block_1[block_row][block_col] = A[A_index];
        } else {// this thread walks off array
            A_block_1[block_row][block_col] = 0;
        }

        if (B_index < square_dim*square_dim) {
            B_block_1[block_row][block_col] = B[B_index];
        } else {// this thread walks off array
            B_block_1[block_row][block_col] = 0;
        }
        __syncthreads();


        //each thread within the block dim loops over rows to add results
        //returning single element in C
#pragma unroll
        for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c += A_block[block_row][k] *  B_block[k][block_col];//
            _c += A_block_1[block_row][k] *  B_block_1[k][block_col];//
        }
        __syncthreads();


    }

    if((row < square_dim) && (col < square_dim)) { // if within matrix bounds
        C[row * square_dim + col] = _c;
    }
}

//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMul_no_unroll(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    __shared__ _DOUBLE_ A_block[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
    __shared__ _DOUBLE_ B_block[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
    int block_row = threadIdx.y;
    int block_col = threadIdx.x;

    //deal with block size != tile size
    int block_x_offset = min(COL_BLOCK_SIZE, blockDim.x);
    int block_y_offset = min(ROW_BLOCK_SIZE, blockDim.y);

    int row =  blockIdx.y*block_y_offset + block_row;
    int col =  blockIdx.x*block_x_offset + block_col;


//    int index = row *square_dim + col;
//    int row = blockIdx.y * blockDim.y + block_col;
    _DOUBLE_ _c = 0;
    int col_offset;
    int row_offset;
    int A_index;
    int B_index;
#pragma unroll
    for (unsigned int kk = 0; kk < gridDim.x; ++kk) {

        col_offset = (kk*block_x_offset + block_col);
        row_offset = (kk*block_y_offset + block_row );
        A_index = row * square_dim + col_offset;
        B_index = row_offset * square_dim + col;

        //load sub-blocks into shared memory: each thread does one load
        if (A_index < square_dim*square_dim) {
            A_block[block_row][block_col] = A[A_index];
        } else {// this thread walks off array
            A_block[block_row][block_col] = 0;
        }

        if (B_index < square_dim*square_dim) {
            B_block[block_row][block_col] = B[B_index];
        } else {// this thread walks off array
            B_block[block_row][block_col] = 0;
        }
        __syncthreads();


        //each thread within the block dim loops over rows to add results
        //returning single element in C
#pragma unroll
        for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c += A_block[block_row][k] *  B_block[k][block_col];//
        }
        __syncthreads();

    }


    if((row < square_dim) && (col < square_dim)) { // if within matrix bounds
        C[row * square_dim + col] = _c;
    }
}

//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMul_naiveSM(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
    __shared__ _DOUBLE_ A_block[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
    __shared__ _DOUBLE_ B_block[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
    int block_row = threadIdx.y;
    int block_col = threadIdx.x;
    int row =  blockIdx.y*ROW_BLOCK_SIZE + block_row;
    int col =  blockIdx.x*COL_BLOCK_SIZE + block_col;

    _DOUBLE_ _c = 0;
    int col_offset;
    int row_offset;
    int A_index;
    int B_index;
#pragma unroll
    for (unsigned int kk = 0; kk < gridDim.x; ++kk) {

        col_offset = (kk*COL_BLOCK_SIZE + block_col);
        row_offset = (kk*ROW_BLOCK_SIZE + block_row );

        A_index = row * square_dim + col_offset;
        B_index = row_offset * square_dim + col;

        //load sub-blocks into shared memory: each thread does one load
        if (A_index < square_dim*square_dim) {
            A_block[block_row][block_col] = A[A_index];
        } else {// this thread walks off array
            A_block[block_row][block_col] = 0;
        }

        if (B_index < square_dim*square_dim) {
            B_block[block_col][block_row] = B[B_index];
        } else {// this thread walks off array
            B_block[block_col][block_row] = 0;
        }
        __syncthreads();


        //each thread within the block dim loops over rows to add results
        //returning single element in C
#pragma unroll
        for (unsigned int k = 0; k < COL_BLOCK_SIZE; ++k) {
            _c += A_block[block_row][k] *  B_block[block_col][k];//
        }
        __syncthreads();

    }


    if((row < square_dim) && (col < square_dim)) { // if within matrix bounds
        C[row * square_dim + col] = _c;
    }
}

__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
    int num_elements = square_dim*square_dim;
    int block_row = threadIdx.y;
    int block_col = threadIdx.x;
    
    //since we are operating on adjacent squares
    int block_index_y = (blockIdx.y*NUM_SIMULTANEOUS_C_COL_ELEMENTS);
    int block_index_x = (blockIdx.x*NUM_SIMULTANEOUS_C_ROW_ELEMENTS);

    __shared__ _DOUBLE_ A_block_0y[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
    __shared__ _DOUBLE_ A_block_1y[ROW_BLOCK_SIZE][COL_BLOCK_SIZE];

    __shared__ _DOUBLE_ B_block_0x[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
    __shared__ _DOUBLE_ B_block_1x[COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
    //deal with block size != tile size
    int block_x_offset = COL_BLOCK_SIZE;//min(COL_BLOCK_SIZE, blockDim.x);//
    int block_y_offset = ROW_BLOCK_SIZE;//min(ROW_BLOCK_SIZE, blockDim.y); //


    int row0y =  (block_index_y)*block_y_offset+ block_row;
    int row1y =  (block_index_y+1)*block_y_offset + block_row;
    int col0x =  (block_index_x)*block_x_offset + block_col;
    int col1x =  (block_index_x+1)*block_x_offset + block_col;
    
    _DOUBLE_ _c0y0x = 0;
    _DOUBLE_ _c0y1x = 0;
    _DOUBLE_ _c1y0x = 0;
    _DOUBLE_ _c1y1x = 0;

    int B_increment = block_y_offset*square_dim;

    //if incrementing inside for loop
    int B_index_0x = col0x + block_row* square_dim  ;
    int B_index_1x = col1x + block_row* square_dim ;

    int A_index_0y = row0y * square_dim + block_col;
    int A_index_1y = row1y * square_dim + block_col ;
//  unsigned int bank_conflict_avoidance_index;
#pragma unroll
    for (unsigned int stride = 0;
         stride < gridDim.x*NUM_SIMULTANEOUS_C_COL_ELEMENTS;
         ++stride
            ,A_index_0y += block_x_offset
            ,A_index_1y += block_x_offset
            ,B_index_0x += B_increment
            ,B_index_1x += B_increment
            ) {

//Note: could probably avoid checks for  a good portion of loop if we
// split it up to do 1 loop over indexes we know will be inbound,
// then this will be the second loop over the rest

////////////load sub-blocks into shared memory: each thread does one load to each array//////////////////

      //check if both A indices inbounds
        if (A_index_1y < num_elements) {
            A_block_0y[block_row][block_col] = A[A_index_0y];
            A_block_1y[block_row][block_col] = A[A_index_1y];
        } else {// this thread walks off array
            A_block_1y[block_row][block_col] = 0;
            //check if A0y direction inbounds
            if (A_index_0y < num_elements) {
                A_block_0y[block_row][block_col] = A[A_index_0y];
            } else {// this thread walks off array
                A_block_0y[block_row][block_col] = 0;
            }
        }

        //check if both B indices inbounds
        if (B_index_1x < num_elements) {
            B_block_0x[block_row][block_col] = B[B_index_0x];
            B_block_1x[block_row][block_col] = B[B_index_1x];
        } else {// this thread walks off array
            B_block_1x[block_row][block_col] = 0;
            //check if B0x index inbounds

            if (B_index_0x < num_elements) {
                B_block_0x[block_row][block_col] = B[B_index_0x];
            } else {// this thread walks off array
                B_block_0x[block_row][block_col] = 0;
            }
        }
        __syncthreads();


//        each thread within the block dim loops over rows to add results


// Note: due to thread divergence, produced marginally better results than having all threads compute

      if((row1y < square_dim) ) { // if within row bound
        if (col1x < square_dim){//all fit
#pragma unroll
          for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c0y0x += A_block_0y[block_row][k] *  B_block_0x[k][block_col];//
            _c0y1x += A_block_0y[block_row][k] *  B_block_1x[k][block_col];//
            _c1y0x += A_block_1y[block_row][k] *  B_block_0x[k][block_col];//
            _c1y1x += A_block_1y[block_row][k] *  B_block_1x[k][block_col];//
          }
        } else if(col0x < square_dim){//if within col bound
#pragma unroll
          for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c0y0x += A_block_0y[block_row][k] *  B_block_0x[k][block_col];//
            _c1y0x += A_block_1y[block_row][k] *  B_block_0x[k][block_col];//
          }
        }
      }
      else if((row0y < square_dim) ) { // if within matrix bounds
        if (col1x < square_dim){//both cols fit
#pragma unroll
          for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c0y0x += A_block_0y[block_row][k] *  B_block_0x[k][block_col];//
            _c0y1x += A_block_0y[block_row][k] *  B_block_1x[k][block_col];//
          }
        } else if(col0x < square_dim){//only 1 col fits
#pragma unroll
          for (unsigned int k = 0; k < block_x_offset; ++k) {
            _c0y0x += A_block_0y[block_row][k] *  B_block_0x[k][block_col];//
          }        }
      }
      __syncthreads();

//        #pragma unroll
//        for (k = 0; k < block_x_offset; ++k) {
//
//            _c0y0x += A_block_0y[block_row][k] *  B_block_0x[k][block_col];//
//            _c0y1x += A_block_0y[block_row][k] *  B_block_1x[k][block_col];//
//            _c1y0x += A_block_1y[block_row][k] *  B_block_0x[k][block_col];//
//            _c1y1x += A_block_1y[block_row][k] *  B_block_1x[k][block_col];//
//        }
//        __syncthreads();

    }

//    if((row0y < square_dim) && (col0x < square_dim)) { // if within matrix bounds
//        C[row0y * square_dim + col0x] = _c0y0x;
//    }
//
//    if((row0y < square_dim) && (col1x < square_dim)) { // if within matrix bounds
//        C[row0y * square_dim + col1x] = _c0y1x;
//    }
//
//    if((row1y < square_dim) && (col0x < square_dim)) { // if within matrix bounds
//        C[row1y * square_dim + col0x] = _c1y0x;
//    }
//
//    if((row1y < square_dim) && (col1x < square_dim)) { // if within matrix bounds
//        C[row1y * square_dim + col1x] = _c1y1x;
//    }

        if((row1y < square_dim) ) { // if within row bound

        if (col1x < square_dim){//all fit
            C[row0y * square_dim + col0x] = _c0y0x;
            C[row0y * square_dim + col1x] = _c0y1x;
            C[row1y * square_dim + col0x] = _c1y0x;
            C[row1y * square_dim + col1x] = _c1y1x;
        } else if(col0x < square_dim){//if within col bound
            C[row0y * square_dim + col0x] = _c0y0x;
            C[row1y * square_dim + col0x] = _c1y0x;
        }
    }
    else if((row0y < square_dim) ) { // if within matrix bounds
        if (col1x < square_dim){//both cols fit
            C[row0y * square_dim + col0x] = _c0y0x;
            C[row0y * square_dim + col1x] = _c0y1x;
        } else if(col0x < square_dim){//only 1 col fits
            C[row0y * square_dim + col0x] = _c0y0x;
        }
    }


    
    
}
//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMul_1sync_element(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
  int num_elements = square_dim*square_dim;
  int block_row = threadIdx.y;
  int block_col = threadIdx.x;
  int nextTile = 0;
  //since we are operating on adjacent squares
  int block_index_y = (blockIdx.y*NUM_SIMULTANEOUS_C_COL_ELEMENTS);
  int block_index_x = (blockIdx.x*NUM_SIMULTANEOUS_C_ROW_ELEMENTS);

  __shared__ _DOUBLE_ A_block_0y[NUM_SM_TILES][ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
  __shared__ _DOUBLE_ A_block_1y[NUM_SM_TILES][ROW_BLOCK_SIZE][COL_BLOCK_SIZE];

  __shared__ _DOUBLE_ B_block_0x[NUM_SM_TILES][COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
  __shared__ _DOUBLE_ B_block_1x[NUM_SM_TILES][COL_BLOCK_SIZE][ROW_BLOCK_SIZE];
  //deal with block size != tile size
  int block_x_offset = min(COL_BLOCK_SIZE, blockDim.x);
  int block_y_offset = min(ROW_BLOCK_SIZE, blockDim.y);


  int row0y =  (block_index_y)*block_y_offset+ block_row;
  int row1y =  (block_index_y+1)*block_y_offset + block_row;
  int col0x =  (block_index_x)*block_x_offset + block_col;
  int col1x =  (block_index_x+1)*block_x_offset + block_col;

  _DOUBLE_ _c0y0x = 0;
  _DOUBLE_ _c0y1x = 0;
  _DOUBLE_ _c1y0x = 0;
  _DOUBLE_ _c1y1x = 0;


  int B_increment = block_y_offset*square_dim;


  //if incrementing inside for loop
  int B_index_0x = col0x + block_row* square_dim ;
  int B_index_1x = col1x + block_row* square_dim ;

  int A_index_0y = row0y * square_dim + block_col ;
  int A_index_1y = row1y * square_dim + block_col ;


  //Load tile for 1st math calculation loop
  if (A_index_1y < num_elements) {
    A_block_0y[nextTile][block_row][block_col] = A[A_index_0y];
    A_block_1y[nextTile][block_row][block_col] = A[A_index_1y];
  } else {// this thread walks off array
    A_block_1y[nextTile][block_row][block_col] = 0;
    //check if A0y direction inbounds
    if (A_index_0y < num_elements) {
      A_block_0y[nextTile][block_row][block_col] = A[A_index_0y];
    } else {// this thread walks off array
      A_block_0y[nextTile][block_row][block_col] = 0;
    }
  }

  //check if both B indices inbounds
  if (B_index_1x < num_elements) {
    B_block_0x[nextTile][block_row][block_col] = B[B_index_0x];
    B_block_1x[nextTile][block_row][block_col] = B[B_index_1x];
  } else {// this thread walks off array
    B_block_1x[nextTile][block_row][block_col] = 0;
    //check if B0x index inbounds

    if (B_index_0x < num_elements) {
      B_block_0x[nextTile][block_row][block_col] = B[B_index_0x];
    } else {// this thread walks off array
      B_block_0x[nextTile][block_row][block_col] = 0;
    }
  }
  A_index_0y += block_x_offset;
  A_index_1y += block_x_offset;
  B_index_0x += B_increment;
  B_index_1x += B_increment;
  __syncthreads();

#pragma unroll
  for (unsigned int stride = 0;
       stride < gridDim.x*NUM_SIMULTANEOUS_C_COL_ELEMENTS -1;
       ++stride
               ,A_index_0y += block_x_offset
               ,A_index_1y += block_x_offset
               ,B_index_0x += B_increment
               ,B_index_1x += B_increment
          ) {
    //each thread within the block dim loops over rows to add results
    //returning single element in C
#pragma unroll
    for (unsigned int k = 0; k < block_x_offset; ++k) {
      _c0y0x += A_block_0y[nextTile][block_row][k] *  B_block_0x[nextTile][k][block_col];//
      _c0y1x += A_block_0y[nextTile][block_row][k] *  B_block_1x[nextTile][k][block_col];//
      _c1y0x += A_block_1y[nextTile][block_row][k] *  B_block_0x[nextTile][k][block_col];//
      _c1y1x += A_block_1y[nextTile][block_row][k] *  B_block_1x[nextTile][k][block_col];//
    }

    //Load the tile for the next calculation
    nextTile = !nextTile;
    if (A_index_1y < num_elements) {
      A_block_0y[nextTile][block_row][block_col] = A[A_index_0y];
      A_block_1y[nextTile][block_row][block_col] = A[A_index_1y];
    } else {// this thread walks off array
      A_block_1y[nextTile][block_row][block_col] = 0;
      //check if A0y direction inbounds
      if (A_index_0y < num_elements) {
        A_block_0y[nextTile][block_row][block_col] = A[A_index_0y];
      } else {// this thread walks off array
        A_block_0y[nextTile][block_row][block_col] = 0;
      }
    }

    //check if both B indices inbounds
    if (B_index_1x < num_elements) {
      B_block_0x[nextTile][block_row][block_col] = B[B_index_0x];
      B_block_1x[nextTile][block_row][block_col] = B[B_index_1x];
    } else {// this thread walks off array
      B_block_1x[nextTile][block_row][block_col] = 0;
      //check if B0x index inbounds

      if (B_index_0x < num_elements) {
        B_block_0x[nextTile][block_row][block_col] = B[B_index_0x];
      } else {// this thread walks off array
        B_block_0x[nextTile][block_row][block_col] = 0;
      }
    }
    __syncthreads();

  }

  //perform final calculation
#pragma unroll
  for (unsigned int k = 0; k < block_x_offset; ++k) {
    _c0y0x += A_block_0y[nextTile][block_row][k] *  B_block_0x[nextTile][k][block_col];//
    _c0y1x += A_block_0y[nextTile][block_row][k] *  B_block_1x[nextTile][k][block_col];//
    _c1y0x += A_block_1y[nextTile][block_row][k] *  B_block_0x[nextTile][k][block_col];//
    _c1y1x += A_block_1y[nextTile][block_row][k] *  B_block_1x[nextTile][k][block_col];//
  }
  __syncthreads();

//perform assignment
  if((row0y < square_dim) && (col0x < square_dim)) { // if within matrix bounds
    C[row0y * square_dim + col0x] = _c0y0x;
  }

  if((row0y < square_dim) && (col1x < square_dim)) { // if within matrix bounds
    C[row0y * square_dim + col1x] = _c0y1x;
  }

  if((row1y < square_dim) && (col0x < square_dim)) { // if within matrix bounds
    C[row1y * square_dim + col0x] = _c1y0x;
  }

  if((row1y < square_dim) && (col1x < square_dim)) { // if within matrix bounds
    C[row1y * square_dim + col1x] = _c1y1x;
  }


}

//__global__ void matMul(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
__global__ void matMul_altSync(int square_dim, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

  int nextTile = 0;


  __shared__ _DOUBLE_ A_block[2][ROW_BLOCK_SIZE][COL_BLOCK_SIZE];
  __shared__ _DOUBLE_ B_block[2][COL_BLOCK_SIZE][ROW_BLOCK_SIZE];


  int block_row = threadIdx.y;
  int block_col = threadIdx.x;
  //deal with block size != tile size
  int block_x_offset = min(COL_BLOCK_SIZE, blockDim.x);
  int block_y_offset = min(ROW_BLOCK_SIZE, blockDim.y);

  int row =  blockIdx.y*block_y_offset + block_row;
  int col =  blockIdx.x*block_x_offset + block_col;
//    int index = row *square_dim + col;
//    int row = blockIdx.y * blockDim.y + block_col;
  _DOUBLE_ _c = 0;

  int col_offset;
  int row_offset;
  int A_index;
  int B_index;
  col_offset = (block_col);
  row_offset = (block_row );
  A_index = row * square_dim + col_offset;
  B_index = row_offset * square_dim + col;

  //load sub-blocks into shared memory: each thread does one load
  if (A_index < square_dim*square_dim) {
    A_block[nextTile][block_row][block_col] = A[A_index];
  } else {// this thread walks off array
    A_block[nextTile][block_row][block_col] = 0;
  }

  if (B_index < square_dim*square_dim) {
    B_block[nextTile][block_row][block_col] = B[B_index];
  } else {// this thread walks off array
    B_block[nextTile][block_row][block_col] = 0;
  }
  __syncthreads();


#pragma unroll
  for (unsigned int kk = 1; kk < gridDim.x; ++kk) {
    //each thread within the block dim loops over rows to add results
    //returning single element in C

#pragma unroll
    for (unsigned int k = 0; k < block_x_offset; ++k) {
      _c += A_block[nextTile][block_row][k] *  B_block[nextTile][k][block_col];//
    }

    //load next tile
    nextTile = !nextTile;
    col_offset = (kk*block_x_offset + block_col);
    row_offset = (kk*block_y_offset + block_row );
    A_index = row * square_dim + col_offset;
    B_index = row_offset * square_dim + col;

    //load sub-blocks into shared memory: each thread does one load
    if (A_index < square_dim*square_dim) {
      A_block[nextTile][block_row][block_col] = A[A_index];
    } else {// this thread walks off array
      A_block[nextTile][block_row][block_col] = 0;
    }

    if (B_index < square_dim*square_dim) {
      B_block[nextTile][block_row][block_col] = B[B_index];
    } else {// this thread walks off array
      B_block[nextTile][block_row][block_col] = 0;
    }
    __syncthreads();

  }
#pragma unroll
  for (unsigned int k = 0; k < block_x_offset; ++k) {
    _c += A_block[nextTile][block_row][k] *  B_block[nextTile][k][block_col];//
  }
  __syncthreads();

  if((row < square_dim) && (col < square_dim)) { // if within matrix bounds
    C[row * square_dim + col] = _c;
  }
}